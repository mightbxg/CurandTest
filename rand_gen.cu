#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cassert>
#include <mutex>
#include <random>
#include <sstream>

#include "rand_gen.h"

#define CUPRINT(x, ...) \
  { printf("\33[33m(CUDA) " x "\n\33[0m", ##__VA_ARGS__); }

#define CUDA_CHECK(err)                                                          \
  do {                                                                           \
    hipError_t err_ = (err);                                                    \
    if (err_ != hipSuccess) {                                                   \
      std::stringstream ss;                                                      \
      ss << "CUDA error " << int(err_) << " at " << __FILE__ << ":" << __LINE__; \
      throw std::runtime_error(ss.str());                                        \
    }                                                                            \
  } while (false)

#define CURAND_CHECK(err)                                                          \
  do {                                                                             \
    hiprandStatus_t err_ = (err);                                                   \
    if (err_ != HIPRAND_STATUS_SUCCESS) {                                           \
      std::stringstream ss;                                                        \
      ss << "cuRAND error " << int(err_) << " at " << __FILE__ << ":" << __LINE__; \
      throw std::runtime_error(ss.str());                                          \
    }                                                                              \
  } while (false)

#define CUDA_CHECK_LAST_ERROR                                                    \
  do {                                                                           \
    hipError_t err_ = hipGetLastError();                                       \
    if (err_ != hipSuccess) {                                                   \
      std::stringstream ss;                                                      \
      ss << "CUDA error " << int(err_) << " at " << __FILE__ << ":" << __LINE__; \
      throw std::runtime_error(ss.str());                                        \
    }                                                                            \
  } while (false)

namespace cu {

void CudaFree(void* dev_ptr) { CUDA_CHECK(hipFree(dev_ptr)); }

void CudaReserveMemory(void** dev_ptr, size_t old_size, size_t new_size) {
  if (new_size > old_size) {
    hipFree(*dev_ptr);
    CUDA_CHECK(hipMalloc(dev_ptr, new_size));
  }
}

void MemcpyToDevice(void* dev_ptr, const void* host_ptr, size_t size) {
  if (size > 0) CUDA_CHECK(hipMemcpy(dev_ptr, host_ptr, size, hipMemcpyHostToDevice));
}

void MemcpyToHost(void* host_ptr, const void* dev_ptr, size_t size) {
  if (size > 0) CUDA_CHECK(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost));
}

__global__ void PrintValues(float* data, size_t num) {
  unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id >= num) return;
  printf("%d: %f %p\n", id, data[id], data + id);
}

void InspectValues(float* dev_ptr, std::size_t num) {
  PrintValues<<<1, num>>>(dev_ptr, num);
  hipDeviceSynchronize();
}

static hipStream_t g_stream;
static void InitStream() {
  static std::once_flag flag;
  std::call_once(flag, [] { CUDA_CHECK(hipStreamCreate(&g_stream)); });
}

static hiprandGenerator_t g_gen;
void HostApiGenerator::Setup(std::size_t num) {
  InitStream();
  std::random_device rd;
  CURAND_CHECK(hiprandCreateGenerator(&g_gen, HIPRAND_RNG_PSEUDO_MT19937));
  CURAND_CHECK(hiprandSetStream(g_gen, g_stream));
  CURAND_CHECK(hiprandSetGeneratorOrdering(g_gen, HIPRAND_ORDERING_PSEUDO_DEFAULT));
  CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(g_gen, rd()));
  num_ = num;
}

void HostApiGenerator::Generate(float* dev_ptr, float mean, float stddev) {
  CURAND_CHECK(hiprandGenerateNormal(g_gen, dev_ptr, num_, mean, stddev));
  hipStreamSynchronize(g_stream);
}

template <int ApiType>
struct GetStateType {};

template <>
struct GetStateType<XORWOW> {
  using type = hiprandState;
};

template <>
struct GetStateType<MRG32k3a> {
  using type = hiprandStateMRG32k3a;
};

template <>
struct GetStateType<Philox4> {
  using type = hiprandStatePhilox4_32_10_t;
};

template <int ApiType>
__global__ void SetupKernel(typename GetStateType<ApiType>::type* state, size_t num) {
  unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id >= num) return;
  hiprand_init(1234, 0, id, state + id);
}

template <int ApiType>
static typename GetStateType<ApiType>::type* g_dev_states;

template <int ApiType>
void DeviceApiGenerator<ApiType>::Setup(std::size_t num) {
  assert((num & 1) == 0);
  InitStream();
  CUDA_CHECK(hipMalloc((void**)&g_dev_states<ApiType>, num * sizeof(typename GetStateType<ApiType>::type)));
  const unsigned int thd_num = 512;
  const unsigned int blk_num = (num + thd_num - 1) / thd_num;
  SetupKernel<ApiType><<<blk_num, thd_num, 0, g_stream>>>(g_dev_states<ApiType>, num);
  hipStreamSynchronize(g_stream);
  num_ = num;
}

template <typename StateType>
__global__ void GenerateNormalKernel(StateType* state, float mean, float stddev, size_t num, float* result) {
  unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id >= num) return;
  auto v = hiprand_normal(state + id);
  result[id] = v * stddev + mean;
}

template <int ApiType>
void DeviceApiGenerator<ApiType>::Generate(float* dev_ptr, float mean, float stddev) {
  const unsigned int thd_num = 512;
  const unsigned int blk_num = (num_ + thd_num - 1) / thd_num;
  GenerateNormalKernel<<<blk_num, thd_num, 0, g_stream>>>(g_dev_states<ApiType>, mean, stddev, num_, dev_ptr);
  hipStreamSynchronize(g_stream);
}

template <int ApiType>
void DeviceApiGenerator<ApiType>::Cleanup() {
  CUDA_CHECK(hipFree(g_dev_states<ApiType>));
}

template class DeviceApiGenerator<XORWOW>;
template class DeviceApiGenerator<MRG32k3a>;
template class DeviceApiGenerator<Philox4>;

}  // namespace cu
